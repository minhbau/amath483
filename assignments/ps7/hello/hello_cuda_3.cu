#include "hip/hip_runtime.h"
#include "Timer.hpp"
#include <iostream>
#include <string>

__global__ void add(int n, const float* x, float* y) {
  int index  = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(int argc, char* argv[]) {

  size_t exponent   = 24;
  size_t num_trips  = 1;
  int    block_size = 256;

  if (argc >= 2) exponent = std::stol(argv[1]);
  if (argc >= 3) num_trips = std::stol(argv[2]);
  if (argc >= 4) block_size = std::stol(argv[3]);

  size_t N = 1 << exponent;

  float *x = nullptr, *y = nullptr;

  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  for (int i = 0; i < N; i++)
    y[i] = 2.0 * (x[i] = 1.0f);

  int num_blocks = (N + block_size - 1) / block_size;

  DEF_TIMER(cuda_call);
  START_TIMER(cuda_call);
  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    add<<<num_blocks, block_size>>>(N, x, y);
    hipDeviceSynchronize();
  }
  double gpu_time = STOP_TIMER_QUIETLY(cuda_call);

  std::cout << exponent << "\t" << num_trips << "\t" << gpu_time << std::endl;

  hipFree(x);
  hipFree(y);
  return 0;
}
