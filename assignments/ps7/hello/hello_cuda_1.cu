#include "hip/hip_runtime.h"
#include "Timer.hpp"
#include <iostream>
#include <string>

__global__ void add(int n, float* x, float* y) {
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(int argc, char* argv[]) {

  size_t exponent  = 24;
  size_t num_trips = 1;

  if (argc >= 2) exponent = std::stol(argv[1]);
  if (argc >= 3) num_trips = std::stol(argv[2]);

  size_t N = 1 << exponent;

  float *x = nullptr, *y = nullptr;

  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  for (int i = 0; i < N; i++)
    y[i] = 2.0 * (x[i] = 1.0f);

  DEF_TIMER(cuda_call);
  START_TIMER(cuda_call);
  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    add<<<1, 1>>>(N, x, y);
    hipDeviceSynchronize();
  }
  double gpu_time = STOP_TIMER_QUIETLY(cuda_call);

  hipFree(x);
  hipFree(y);

  std::cout << exponent << "\t" << num_trips << "\t" << gpu_time << std::endl;

  return 0;
}
