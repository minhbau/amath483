#include "hip/hip_runtime.h"
#include "Timer.hpp"
#include <iostream>
#include <string>

__global__ void add(int n, float* x, float* y) {
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(int argc, char* argv[]) {

  size_t exponent  = 24;
  size_t num_trips = 1;

  if (argc >= 2) exponent = std::stol(argv[1]);
  if (argc >= 3) num_trips = std::stol(argv[2]);

  size_t N = 1 << exponent;

  float *x = nullptr, *y = nullptr;

  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  for (int i = 0; i < N; i++)
    y[i] = 2.0 * (x[i] = 1.0f);

  DEF_TIMER(cuda_call);
  START_TIMER(cuda_call);
  for (size_t i = 0; i < num_trips; ++i) {
    add<<<1, 1>>>(N, x, y);
  }
  STOP_TIMER(cuda_call);

  hipDeviceSynchronize();

  hipFree(x);
  hipFree(y);

  return 0;
}
