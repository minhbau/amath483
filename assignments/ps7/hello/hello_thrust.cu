#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include "Timer.hpp"
#include <iostream>
#include <string>

template<typename T>
struct add {
  __host__ __device__ T operator()(const T& x, const T& y) const { return x + y + .15; }
};

int main(int argc, char* argv[]) {

  size_t exponent  = 24;
  size_t num_trips = 1;

  if (argc >= 2) exponent = std::stol(argv[1]);
  if (argc >= 3) num_trips = std::stol(argv[2]);

  size_t N = 1 << exponent;

  thrust::host_vector<float>   x(N), y(N);
  thrust::device_vector<float> X(N), Y(N);

  for (int i = 0; i < N; i++)
    y[i] = 2.0 * (x[i] = 1.0f);

  thrust::copy(x.begin(), x.end(), X.begin());
  thrust::copy(y.begin(), y.end(), Y.begin());

  DEF_TIMER(cuda_call);
  START_TIMER(cuda_call);
  for (size_t i = 0; i < num_trips; ++i) {
    hipDeviceSynchronize();
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), thrust::plus<float>());
  }
  hipDeviceSynchronize();
  thrust::copy(Y.begin(), Y.end(), y.begin());

  double gpu_time = STOP_TIMER_QUIETLY(cuda_call);

  std::cout << exponent << "\t" << num_trips << "\t" << gpu_time << std::endl;

  return 0;
}
