#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include "Timer.hpp"
#include <iostream>
#include <string>
#include <cmath>
#include <algorithm>
#include <numeric>
#include <random>


template <typename VectorType, typename T>
void randomize(VectorType &x, T scale) {
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution(-scale, scale);
  static auto dice = std::bind(distribution, generator);

  for (size_t i = 0; i < x.size(); ++i) {
    x[i] = dice();
  }
}


template <typename T>
struct square
{
	/* write me */
};


int main(int argc, char* argv[]) {

  size_t exponent           = 27;
  size_t num_trips          = 1;

  if (argc >= 2) exponent   = std::stol(argv[1]);
  if (argc >= 3) num_trips  = std::stol(argv[2]);

  size_t num_elements = 1 << exponent;
  
  thrust::host_vector<float> x(num_elements);
  randomize(x, 10.0f);
  thrust::device_vector<float> X(num_elements);
  thrust::copy(x.begin(), x.end(), X.begin());

  float                init = 0.0;
  square<float>        unary_op;
  thrust::plus<float> binary_op;

  DEF_TIMER(gpu_norm);
  START_TIMER(gpu_norm);

  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    /* write me -- use transform reduce (?) with unary op and binary op defined above */

    hipDeviceSynchronize();
  }

  double cuda_time = STOP_TIMER_QUIETLY(gpu_norm);
  std::cout << exponent << "\t" << num_trips << "\t" << cuda_time << std::endl;

  return 0;
}
