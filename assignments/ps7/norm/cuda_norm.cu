#include "hip/hip_runtime.h"
#include "Timer.hpp"
#include <iostream>


__global__
void sq(int n, float *x, float *y){
     /* Write me -- kernel */
}


int main(int argc, char* argv[]) {

  size_t exponent           = 27;
  size_t num_trips          = 1;

  if (argc >= 2) exponent   = std::stol(argv[1]);
  if (argc >= 3) num_trips  = std::stol(argv[2]);

  size_t N = 1 << exponent;
  int block_size = 256;
  int num_blocks = (N + block_size - 1) / block_size;

  float *x = nullptr, *y = nullptr;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, num_blocks*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = i;
  }

  DEF_TIMER(cuda_norm);
  START_TIMER(cuda_norm);
  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    /* write me -- launch sq kernel */
    hipDeviceSynchronize();
  }
  /* write me: final step, copy out values from y and add on cpu */

  double cuda_time = STOP_TIMER_QUIETLY(cuda_norm);
  std::cout << exponent << "\t" << num_trips << "\t" << cuda_time << std::endl;

  hipFree(x); hipFree(y);
  
  return 0;
}
